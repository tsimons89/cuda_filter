#include "hip/hip_runtime.h"
#include "gpu_filter.h"
#include <ctime>
#include <stdio.h>

__global__ void filter( float *in, float *out, int cols,int rows) {
	 int x = blockIdx.x * blockDim.x + threadIdx.x; 
 	 int y = blockIdx.y * blockDim.y + threadIdx.y;

 	 if(x >= cols || y >= rows)
 	 	return;
 	
 	 float res = (in[y*rows + x] - in[y*rows + x + 2]);
 	 //out[y*rows +x] = in[y*rows + x];
 	 out[y*rows +x] = res;
}

void upload_to_gpu(float** cpu_src, float** gpu_dst,float cols,float rows){
	clock_t begin = clock();
	hipMalloc((void**)gpu_dst, cols * rows * sizeof(float));
	hipMemcpy( *gpu_dst, *cpu_src, cols * rows * sizeof(float),hipMemcpyHostToDevice);
	clock_t end = clock();
	printf("Upload time: %d\n",end-begin);
}

void downloadfrom_gpu(float** gpu_src, float** cpu_dst,float cols,float rows){
	//cpu_dst = (float**) malloc(cols * rows * sizeof(float));
	clock_t begin = clock();
	hipMemcpy(&cpu_dst,gpu_src,cols * rows * sizeof(float),hipMemcpyDeviceToHost);
	clock_t end = clock();
	printf("Download time: %d\n",end-begin);
}


void gpu_filter(float * input,float * output,int cols,int rows){
	clock_t total_b = clock();
	float *in_gpu,*out_gpu;
	clock_t mal_b = clock();
	hipMalloc((void**)&out_gpu, cols * rows * sizeof(float));
	clock_t mal_e = clock();
	printf("Mal time: %d\n",mal_e - mal_b);
	upload_to_gpu(&input,&in_gpu,cols,rows);
	dim3 blocks(cols/16,rows/16);
	dim3 threads(16,16);
	clock_t begin = clock();
	filter<<<blocks,threads>>>(in_gpu,out_gpu,cols,rows);
	clock_t end = clock();

	printf("Add time: %d\n",end-begin);


	 begin = clock();
	hipMemcpy(output,out_gpu,rows*cols*sizeof(float),hipMemcpyDeviceToHost);
	 end = clock();
	printf("Download time: %d\n",end-begin);
	clock_t total_e = clock();
	printf("Total time: %d\n",total_e - total_b);
	// downloadfrom_gpu(&out_gpu,&output,cols,rows);
	hipFree( in_gpu );
	hipFree( out_gpu );
}
