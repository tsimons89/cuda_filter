#include "hip/hip_runtime.h"
#include <opencv2/opencv.hpp>
#include "gpu_filter.h"
#include <ctime>
#include <stdio.h>
using namespace cv;
#define N 10
int main( void ) {
	Mat test_mat = imread("image.jpg",IMREAD_GRAYSCALE);
	test_mat.convertTo(test_mat,CV_32F);
	float * image,*output;
	output = (float*)malloc(test_mat.total()*sizeof(float));
	image = (float*)test_mat.data;
	clock_t begin = clock();
	gpu_filter(image,output,test_mat.cols,test_mat.rows);
	clock_t end = clock();
	printf("Cycles: %d\n",end-begin);
	begin = clock();
	gpu_filter(image,output,test_mat.cols,test_mat.rows);
	end = clock();
	printf("Cycles: %d\n",end-begin);
	// Mat out_mat(test_mat.size(),CV_32F,output);

	// test_mat.convertTo(test_mat,CV_8U);
	// out_mat.convertTo(out_mat,CV_8U);
	// imshow("in",test_mat);
	// imshow("out",out_mat);
	// waitKey(0);

	begin = clock();
	Sobel(test_mat,test_mat,-1,1,0);
	end = clock();
	printf("Cycles: %d\n",end-begin);

	begin = clock();
	Sobel(test_mat,test_mat,-1,1,0);
	end = clock();
	printf("Cycles: %d\n",end-begin);


	return 0;
}
